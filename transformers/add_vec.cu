#include "hip/hip_runtime.h"
#include "add_vec.hpp"
#include <hip/hip_runtime.h>
#include <cassert>

__global__ void add_vec_kernel(const int length, float* da, float* db, float* dc)
{
	int tix = threadIdx.x;

	if (tix < length)
	{ 
		dc[tix] = da[tix] + db[tix];
	}
}

void add_vectors(
	const int length,
	const Vector& ha,
	const Vector& hb,
	Vector& hc)
{
	assert(length <= 1024);

	// allocate memories on the device
	int buffer_size = length * sizeof(float);

	float *da, *db, *dc;
	hipMalloc(&da, buffer_size);
	hipMalloc(&db, buffer_size);
	hipMalloc(&dc, buffer_size);

	// move memory from host to device
	hipMemcpy(da, ha.data(), buffer_size, hipMemcpyHostToDevice);
	hipMemcpy(db, hb.data(), buffer_size, hipMemcpyHostToDevice);

	// execute kernel
	dim3 gs(1, 1, 1);
	dim3 bs(1024, 1, 1);
	add_vec_kernel<<<gs, bs>>>(length, da, db, dc);

	// move data back
	hipMemcpy(hc.data(), dc, buffer_size, hipMemcpyDeviceToHost);

	// clean up
	hipFree(da);
	hipFree(db);
	hipFree(dc);
}
