#include "hip/hip_runtime.h"
#include "fast_mm.cuh"

constexpr int WARP_SIZE = 32;
constexpr int NUM_WARPS = 8;

constexpr int TS = 64;  // tile size
constexpr int NR = TS / NUM_WARPS;
constexpr int NM = TS / 16;


__device__ void calc_perm_indices(
	const int si,  // source
	const int sj,
	int& ti,       // target
	int& tj) 
{
	int li = si;
	int lj = sj / 16;
	
	int li_1 = li % 16;
	int li_0 = li - li_1;
	int lj_1 = lj;

	int li_prime = lj_1 * 4 + li_1 / 4 + li_0;
	int lj_prime = li_1 % 4;

	ti = li_prime;
	tj = lj_prime * 16 + sj % 16;
}


__global__ void cu_fast_mm_f16_v3_2_kernel(
	const int x_width,
	const int w_width,
	const float16* dx,
	const float16* dw,
	float16* dy)
{
	// output tile
	__shared__ float16 shrd_y[TS * TS];
	for (int ix = 0; ix < TS * TS; ++ix)
	{
		shrd_y[ix] = 0.f;
	}

	// iterate over the shared tiles
	int NT = x_width / TS;
	for (int tk = 0; tk < NT; ++tk)
	{
		__shared__ float16 shrd_x[TS * TS];
		__shared__ float16 shrd_w[TS * TS];

		// load global to shared (xt)
		for (int r = 0; r < NR; ++r)
		{
			int glob_idx = (blockIdx.y * TS + threadIdx.y * NR + r) * x_width + tk * TS + threadIdx.x * 2;

			int ti, tj;
			calc_perm_indices(threadIdx.y * NR + r, threadIdx.x * 2, ti, tj);
			int shrd_idx = ti * TS + tj;

			const int32* wide_glob_x = reinterpret_cast<const int32*>(dx);
			int32* wide_shrd_x = reinterpret_cast<int32*>(shrd_x);

			wide_shrd_x[shrd_idx / 2] = wide_glob_x[glob_idx / 2];
		}

		// load global to shared (wt)
		for (int r = 0; r < NR; ++r)
		{
			int glob_idx = (tk * TS + threadIdx.y * NR + r) * w_width + blockIdx.x * TS + threadIdx.x * 2;

			int ti, tj;
			calc_perm_indices(threadIdx.y * NR + r, threadIdx.x * 2, ti, tj);
			int shrd_idx = ti * TS + tj;

			const int32* wide_glob_w = reinterpret_cast<const int32*>(dw);
			int32* wide_shrd_w = reinterpret_cast<int32*>(shrd_w);

			wide_shrd_w[shrd_idx / 2] = wide_glob_w[glob_idx / 2];
		}

		__syncthreads();

		// wgemm based matmul
		constexpr int width = NM * NM / NUM_WARPS;
		int start = (threadIdx.y % 2) * width;
		int row = threadIdx.y / 2;

        #pragma unroll
		for (int c = 0; c < width; ++c)
		{
			int col = c + start;

			nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, float16> acc;
			nvcuda::wmma::load_matrix_sync(acc, shrd_y + row * TS * 16 + col * TS * 4, 16, nvcuda::wmma::mem_row_major);

            #pragma unroll
			for (int k = 0; k < NM; ++k)
			{
				nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, float16, nvcuda::wmma::row_major> reg_x;
				nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, float16, nvcuda::wmma::row_major> reg_w;

				nvcuda::wmma::load_matrix_sync(reg_x, shrd_x + row * TS * 16 + k * TS * 4, 16);
				nvcuda::wmma::load_matrix_sync(reg_w, shrd_w + k * TS * 16 + col * TS * 4, 16);

				nvcuda::wmma::mma_sync(acc, reg_x, reg_w, acc);
			}

			nvcuda::wmma::store_matrix_sync(shrd_y + row * TS * 16 + col * TS * 4, acc, 16, nvcuda::wmma::mem_row_major);
		}

		__syncthreads();
	}

	// store from shared to global
	for (int r = 0; r < NR; ++r)
	{
		int glob_idx = (blockIdx.y * TS + threadIdx.y * NR + r) * w_width + blockIdx.x * TS + threadIdx.x * 2;
		int ti, tj;
		calc_perm_indices(threadIdx.y * NR + r, threadIdx.x * 2, ti, tj);
		int shrd_idx = ti * TS + tj;

		int32* wide_glob_y = reinterpret_cast<int32*>(dy);
		int32* wide_shrd_y = reinterpret_cast<int32*>(shrd_y);

		wide_glob_y[glob_idx / 2] = wide_shrd_y[shrd_idx / 2];
	}
}


void cu_fast_mm_f16_v3_2(
	const Tensor<float16, CUDA>& xt,  // input (multiple of 128)
	const Tensor<float16, CUDA>& wt,  // weight (multiple of 128)
	Tensor<float16, CUDA>& yt)
{
	const int x_height = xt.shape[0];
	const int x_width = xt.shape[1];
	const int w_width = wt.shape[1];

	const float16* dx = xt.buffer();
	const float16* dw = wt.buffer();
	float16* dy = yt.buffer();

	dim3 bs = { WARP_SIZE, NUM_WARPS, 1 };
	dim3 gs = { calc_req_num_blocks(w_width, TS), calc_req_num_blocks(x_height, TS), 1 };

	cu_fast_mm_f16_v3_2_kernel<<<gs, bs>>>(x_width, w_width, dx, dw, dy);
	CUDA_CHECK_LAST_ERROR();
}
